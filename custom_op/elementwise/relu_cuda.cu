#include "hip/hip_runtime.h"
#include "paddle/extension.h"

template <typename data_t>
__global__ void relu_cuda_forward_kernel(const data_t* x,
                                         data_t* y,
                                         int64_t num) {
  int64_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int64_t i = gid; i < num; i += blockDim.x * gridDim.x) {
    y[i] = max(x[i], static_cast<data_t>(0.));
  }
}

template <typename data_t>
__global__ void relu_cuda_backward_kernel(const data_t* dy,
                                          const data_t* y,
                                          data_t* dx,
                                          int64_t num) {
  int64_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int64_t i = gid; i < num; i += blockDim.x * gridDim.x) {
    dx[i] = dy[i] * (y[i] > 0 ? 1. : 0.);
  }
}

std::vector<paddle::Tensor> relu_cuda_forward(const paddle::Tensor& x) {
  auto out = paddle::empty_like(x);

  int64_t numel = x.numel();
  int64_t block = 512;
  int64_t grid = (numel + block - 1) / block;
  PD_DISPATCH_FLOATING_TYPES(
      x.type(), "relu_cuda_forward_kernel", ([&] {
        relu_cuda_forward_kernel<data_t><<<grid, block, 0, x.stream()>>>(
            x.data<data_t>(), out.data<data_t>(), numel);
      }));

  return {out};
}

std::vector<paddle::Tensor> relu_cuda_backward(const paddle::Tensor& x,
                                               const paddle::Tensor& out,
                                               const paddle::Tensor& grad_out) {
  auto grad_x = paddle::empty_like(x);

  int64_t numel = out.numel();
  int64_t block = 512;
  int64_t grid = (numel + block - 1) / block;
  PD_DISPATCH_FLOATING_TYPES(
      out.type(), "relu_cuda_backward_kernel", ([&] {
        relu_cuda_backward_kernel<data_t><<<grid, block, 0, x.stream()>>>(
            grad_out.data<data_t>(),
            out.data<data_t>(),
            grad_x.data<data_t>(),
            numel);
      }));

  return {grad_x};
}

